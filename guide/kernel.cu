#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <ctime>

#define BLOCK_SIZE 256

__global__ void Sieve(bool * d_inout, int Prime){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	/*          With Divergence          */
	if(((2*tid+1)%Prime) == 0){
		d_inout[tid] = 1;
	}
	/**/
	/*      Without Divergence          *//*
	d_inout[tid] = !(!(1 & (((2*tid+1)%Prime)==0)) & !(1 & d_inout[tid]));
	/**/
}

int main(){
	bool * h_inout, * d_inout;

	//-------------- Taking the range from User --------------//
	int range = 0;
	std::cin >> range;
	int r = sqrt((double)range);

	//--------- Device launch configuration ---------//* Calculating it here to eleminate the time it takes from profiling as well as using its data for memory padding */
	int block_size = BLOCK_SIZE;
	int num_blocks = range/block_size + ( range % block_size == 0 ? 0 : 1);
	dim3 globalDim(num_blocks/2 + num_blocks%2);
	dim3 blockDim(block_size);
	
	// ------ Start of Timing -------- //
	clock_t t = clock();

	//-------------- Alocating and Initializing memory in host & device ------------//
	hipHostAlloc(&h_inout,sizeof(bool)*num_blocks*BLOCK_SIZE/2,hipHostMallocDefault);
	hipMalloc(&d_inout,num_blocks*BLOCK_SIZE*sizeof(bool)/2);
	hipMemcpy(d_inout,h_inout,range*sizeof(bool)/2,hipMemcpyHostToDevice);

	//-------------- Finding all prime numbers between 2 and sqrt(range) -------------//
	bool * Integers = (bool *) calloc(sizeof(bool),r/2);
	int * Primes = (int *) malloc(sizeof(int)*r);
	int num_primes = 1;
	for(int i = 3; i*i < r; i+=2){
		if(Integers[i/2] == 0){
			for(int j = 3*i; j < r; j+=2*i){
				Integers[j/2] = 1;
			}
		}
	}
	
	// ------- Counting the nmber of primes below sqrt(range) ----------//
	for(int i = 3; i < r; i+=2)
		if(Integers[i/2] == 0)
			num_primes += 1;

	Primes = (int *) malloc(sizeof(int)*num_primes);
	Primes[0] = 2;
	for(int i = 3, j = 1; i < r; i+=2){
		if(Integers[i/2] == 0){
			Primes[j++] = i;
		}
	}

	for(int i = 1; i < num_primes ; i++){
		Sieve<<<globalDim,blockDim>>>(d_inout,Primes[i]);
	}

	hipMemcpy(h_inout,d_inout,range*sizeof(bool)/2,hipMemcpyDeviceToHost);

	// ------ End of Timing ------//
	hipDeviceSynchronize();
	t = clock() - t;

	int count = num_primes;
	for(int i = 3; i < range; i+=2){
		if(h_inout[i/2] == 0){
			count += 1;
		}
	}

	std::cout << count << " ,Primes in " << ((float)t)/CLOCKS_PER_SEC << " Second(s)." << std::endl;

	free(Integers);
	free(Primes);
	hipHostFree(h_inout);
	hipFree(d_inout);

	return EXIT_SUCCESS;
}